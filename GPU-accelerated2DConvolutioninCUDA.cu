#include "hip/hip_runtime.h"
/**
*   CS6023: GPU Programming 
*   Assignment 2
*   
*   Please don't change any existing code in this file.
*
*   Please add necessary memory APIs for your implementation. Use hipFree() 
*   to free up memory as soon as you're done with an allocation. 
*   This will ensure that you don't run out of memory while running
*   large test cases. Use the minimum required memory for your 
*   implementation. DO NOT change the kernel configuration parameters.
*/

#include <chrono>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

using namespace std;

using std::cin;
using std::cout;

typedef long long ll;

 //kernel for convolution operation
__global__ void convolutionKernel(long int* d_mat, long int* d_filter, long int* d_ans, long int m, long int n, long int k) {

    extern __shared__ long int shared_filter[]; //dynamic shared memory for optimization 

    // Loading filter into shared memory
    if (threadIdx.x == 0)
    {
        for (long int i = 0; i < k * k; i++)
        {
            shared_filter[i] = d_filter[i]; //original filter is copied in shared memory
        }
    }

    __syncthreads();
    long int id = blockIdx.x * blockDim.x + threadIdx.x; //Id for particular thread

    long int row = id / n; // Each block corresponds to one row of the output matrix
    long int col = id % n; // Each thread corresponds to one column of the output matrix


    long int padding = k / 2; //padding calculation 
    long int sum = 0;


    for (long int a = -padding; a <= padding; a++) {
        for (long int b = -padding; b <= padding; b++) {

            long int nx = row + a;
            long int ny = col + b;

            if (nx >= 0 && nx < m && ny >= 0 && ny < n) {
                sum += d_mat[nx * n + ny] * shared_filter[(a + padding) * k + (b + padding)];
            }

        }
    }

    d_ans[row * n + col] = sum; //will store final answer 

}

int main(int argc, char** argv) {

    int m,n,k;
    cin>>m>>n>>k;


    long int* h_mat = new long int[m * n];
    long int* h_filter = new long int[k * k];

    long int* h_ans = new long int[m * n];


    for (long int i = 0; i < m * n; i++) {
        cin>>h_mat[i];
    }

    for (long int i = 0; i < k * k; i++) {
        cin>>h_filter[i];
    }

    /**
     * 
     * DO NOT CHANGE ANYTHING ABOVE THIS LINE
     * 
    **/

    /****************************************************Start Here***********************************************************/
    
     //Allocated device memory
    long int* d_mat, * d_filter, * d_ans;
    hipMalloc(&d_mat, m * n * sizeof(long int));
    hipMalloc(&d_filter, k * k * sizeof(long int));
    hipMalloc(&d_ans, m * n * sizeof(long int));

    //copy data from host to device 
    hipMemcpy(d_mat, h_mat, m * n * sizeof(long int), hipMemcpyHostToDevice);
    hipMemcpy(d_filter, h_filter, k * k * sizeof(long int), hipMemcpyHostToDevice);

    //Define block and grid dimentions

    auto start = std::chrono::high_resolution_clock::now();//keep it just before the kernel launch
     
    // Launch kernel
    convolutionKernel << <m, n, k* k * sizeof(long int) >> > (d_mat, d_filter, d_ans, m, n, k);


    auto end = std::chrono::high_resolution_clock::now();//keep it just after the kernel launch
    
    //copy result back to host from device
    hipMemcpy(h_ans, d_ans, m * n * sizeof(long int), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    
    /*$$$$$$$$$$$$$$$$$$$$$$$$Make sure your final output from the device is stored in h_ans.$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$*/

    hipFree(d_mat);
    hipFree(d_filter);
    hipFree(d_ans);
    std::chrono::duration<double> elapsed1 = end - start;
    /**
     * 
     * DO NOT CHANGE ANYTHING BELOW THIS LINE
     * 
    */


    
    std::ofstream file("cuda.out");
    if (file.is_open()) {
        for (long int i = 0; i < m; i++) {
            for (long int j = 0; j < n; j++) {
                file << h_ans[i * n + j] << " ";
            }
            file << "\n";
        }
        file.close();
    } else {
        std::cout << "Unable to open file";
    }

    std::ofstream file2("cuda_timing.out");
    if(file2.is_open()) {
        file2 << elapsed1.count() << "\n";
        file2.close();
    } else {
        std::cout << "Unable to open file";
    }

    return 0;
}